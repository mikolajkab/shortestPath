#include "hip/hip_runtime.h"
// A C++ program for Bellman-Ford's queue-based single source 
// shortest path algorithm. 
#include <bits/stdc++.h>
#include <chrono>
#include <fstream>
#include <omp.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

using namespace std;
using namespace std::chrono;

const string fin_str = "../../../matlab/gr_10000_100.csv";

typedef pair<int, int> iPair; 

// This class represents a directed graph
class Graph 
{ 
public:
	Graph();

	void addEdge(int u, int v, int w);
	
	vector<vector<iPair> > nodes; 
}; 

Graph::Graph() 
{ 
} 

void Graph::addEdge(int u, int v, int w)
{ 
	if (u >= nodes.size())
	{
		nodes.resize(u+1);
	}
	if (v >= nodes.size())
	{
		nodes.resize(v+1);
	}

	nodes[u].push_back(make_pair(v, w)); 
	nodes[v].push_back(make_pair(u, w)); 
} 

__global__ void relax_initial(int * d_dist, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

	d_dist[i] = INF;

	if (i == 0) 
	{
		d_dist[i] = 0;
	}
	__syncthreads();
}

__global__ void bf(int u, int const* d_weights, int* d_dist, int* in_queue, int* came_from)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

	int weight = d_weights[i];
	if (weight < INF)
	{
		if (d_dist[u] + weight < d_dist[v])
		{
			d_dist[v] = d_dist[u] + weight;
			in_queue[v] = true;
			came_from[v] = u;
		}
	}
}

// The main function that finds shortest distances
void BellmanFord(int src, int goal, int n, int h_weights[]) 
{ 
	int threadsPerBlock = 256;
	int blocksPerGrid = ((n + threadsPerBlock.x - 1) / threadsPerBlock.x);
	
	// host 
	int *h_dist = (int *)calloc(sizeof(int), n);
	int *h_in_queue = (int *)calloc(sizeof(bool), n);
	int *h_came_from = (int *)calloc(sizeof(int), n);

	int h_dist[] = {[0 ... n-1] = INT_MAX};
	int h_in_queue[] = {[0 ... n-1] = false};
	int h_came_from[] = {[0 ... n-1] = INT_MAX};

	h_dist[src] = 0;
	h_in_queue[src] = true;
	h_came_from[src] = src;

	// device
	int* d_weights;
	int* d_dist;
	int* d_came_from;
	bool* d_in_queue;

	hipMalloc(&d_weights, n * n * sizeof(int));
	hipMalloc(&d_dist, n * sizeof(int));
	hipMalloc(&d_came_from, sizeof(int));
	hipMalloc(&d_in_queue, sizeof(bool));

	// copy host to device
	hipMemcpy(d_weights, h_weights, n * n * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_dist, h_dist, n * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_came_from, h_came_from, n * sizeof(int), hipMemcpyHostToDevice);

	queue<int> node_queue;
	node_queue.push(src);

	// main loop
	auto start = high_resolution_clock::now();
	while(!node_queue.empty())
	{
		int u = node_queue.front();
		node_queue.pop();
		h_in_queue[u] = false;

		hipMemcpy(d_in_queue, h_in_queue, n * sizeof(int), hipMemcpyHostToDevice);

        // invoke kernel
		bf <<<blocksPerGrid, threadsPerBlock>>>(u, d_weights, d_dist, d_in_queue, d_came_from);
	
		hipMemcpy(h_in_queue, d_in_queue, n * sizeof(int), hipMemcpyDeviceToHost);
	}
	auto stop = high_resolution_clock::now(); 

	hipMemcpy(h_dist, d_dist, sizeof(int) * n, hipMemcpyDeviceToHost);



	// Print shortest distances stored in dist[] 
	ofstream myfile ("bfq.txt");
  	if (myfile.is_open())
  	{
		for (int i = 0; i < graph->nodes.size(); ++i) 
			myfile << i << "\t\t" << dist[i] <<"\n"; 
    	myfile.close();
  	}
  	else cout << "Unable to open file";

	ofstream myfile_path ("bfq_path.txt");
	if (myfile_path.is_open())
	{
		vector<int> path;
		int current = goal;
		while(current != src)
		{
			path.push_back(current);
			current = came_from[current];
		}
		path.push_back(src);
		reverse(path.begin(), path.end());

		for (vector<int>::iterator i = path.begin(); i < path.end(); ++i)
		{
			myfile_path << *i << "\t\t";
		}
    	myfile_path.close();
	} 
  	else cout << "Unable to open file";

	auto duration = duration_cast<milliseconds>(stop - start);
	cout << "duration :" << duration.count() << endl;
} 

//translate 2-dimension coordinate to 1-dimension
int convert_dimension_2D_1D(int x, int y, int n) {
	return x * n + y;
}

int* create_weights(int weights[], N)
{
	int adj_mat[N][N];

	fstream fin;
	fin.open(fin_str, ios::in);

	vector<int> row;
	string line, word;
	getline(fin,line);

	while (!fin.eof())
	{
		row.clear();
		getline(fin, line);
		stringstream s(line);

		while (getline(s, word, ','))  
		{
			row.push_back(stoi(word));
		}
		adj_mat[row[0]-1, row[1]-1] = row[2];
	}
	fin.close();

	for (int i = 0; i < N; i++) 
	{
		for (int j = 0; j < N; j++) 
		{
			inputf >> weights[convert_dimension_2D_1D(i, j, N)];
		}
	}

	return weights;
}



// Driver program to test above functions 
int main()
{
	int N = 10;
	int* mat = (int *)malloc(N * N * sizeof(int));

	int* weights = create_weights(mat, N);

	BellmanFord(0, 10);

	return 0; 
} 
