// A C++ program for Bellman-Ford's single source 
// shortest path algorithm. 
#include <bits/stdc++.h>
#include <chrono>
#include <fstream>
#include <omp.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

using namespace std;
using namespace std::chrono;

#define INF 2000000000

const string fin_str = "../matlab/gr_optimal_control_3rd_order.csv";

__global__ void bf(int n, int const* d_weights, int* d_dist, bool* d_has_change, int* came_from)
{
	int v = blockIdx.x * blockDim.x + threadIdx.x;

	if (v == 0)
		*d_has_change = false;
	__syncthreads();

	if(v >= n)
	{
		return;
	}

	for (int u = 0; u < n; ++u)
	{
		if (u != v)
		{
			int weight = d_weights[u * n + v];
			if (weight < INF)
			{
				if (d_dist[v] > d_dist[u] + weight)
				{
					d_dist[v] = d_dist[u] + weight;
					*d_has_change = true;
					came_from[v] = u;
				}
			}
		}
	}
	__syncthreads();
}

void bf_func(int n, int const* d_weights, int* d_dist, bool* d_has_change, int* came_from)
{
	*d_has_change = false;

	for (int v = 0; v < n; ++v)
	{
		for (int u = 0; u < n; ++u)
		{
			if (u != v)
			{
				int weight = d_weights[u * n + v];
				if (weight < INF)
				{
					if (d_dist[v] > d_dist[u] + weight)
					{
						d_dist[v] = d_dist[u] + weight;
						*d_has_change = true;
						came_from[v] = u;
						cout << "v: " << v << ", u: " << u << ", dist_v: " << d_dist[v] << ", dist_u: " << d_dist[u] << ", weight: " << weight << "\n";
					}
				}
			}
		}
	}
}

//translate 2-dimension coordinate to 1-dimension
int convert_dimension_2D_1D(int x, int y, int n) 
{
	return x * n + y;
}

// The main function that finds shortest distances
void BellmanFord(int src, int goal, int n, int h_weights[]) 
{ 
	dim3 threadsPerBlock = 256;
	dim3 blocksPerGrid = ((n + threadsPerBlock.x - 1) / threadsPerBlock.x);
	
	// host 
	int *h_dist = (int *)calloc(sizeof(int), n);
	int *h_came_from = (int *)calloc(sizeof(int), n);
	bool *h_has_change = (bool *)calloc(sizeof(bool), 1);
	
	for (int i=0; i<n; i++)
	{
		h_dist[i] = INF;
		h_came_from[i] = INF;
	}

	h_dist[src] = 0;
	h_came_from[src] = src;

	// device
	int* d_weights;
	int* d_dist;
	int* d_came_from;
	bool* d_has_change;

	hipMalloc(&d_weights, n * n * sizeof(int));
	hipMalloc(&d_dist, n * sizeof(int));
	hipMalloc(&d_came_from, n * sizeof(int));
	hipMalloc(&d_has_change, sizeof(bool));

	// copy host to device
	hipMemcpy(d_weights, h_weights, n * n * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_dist, h_dist, n * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_came_from, h_came_from, n * sizeof(int), hipMemcpyHostToDevice);

	// for(int i=0; i<n; i++)
	// {
	// 	cout << h_dist[i] << " ";
	// }
	// cout << "\n";

	int counter = 0;
	// main loop
	auto start = high_resolution_clock::now();
	while(true)
	{
		counter++;

        // invoke kernel
		bf <<<blocksPerGrid, threadsPerBlock>>>(n, d_weights, d_dist, d_has_change, d_came_from);
	
		// bf_func(n, h_weights, h_dist, h_has_change, h_came_from);

		hipMemcpy(h_has_change, d_has_change, sizeof(bool), hipMemcpyDeviceToHost);

		if(!(*h_has_change))
		{
			break;
		}
	}

	auto stop = high_resolution_clock::now(); 

	cout << "counter: " << counter << "\n";

	hipMemcpy(h_dist, d_dist, n * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(h_came_from, d_came_from, n * sizeof(int), hipMemcpyDeviceToHost);

	// for(int i=0; i<n; i++)
	// {
	// 	cout << h_dist[i] << " ";
	// }
	// cout << "\n";
	// for(int i=0; i<n; i++)
	// {
	// 	cout << h_came_from[i] << " ";
	// }
	// cout << "\n";

	hipFree(d_weights);
	hipFree(d_dist);
	hipFree(d_came_from);
	hipFree(d_has_change);

	// Print shortest distances stored in dist[] 
	ofstream myfile ("bf.txt");
  	if (myfile.is_open())
  	{
		for (int i = 0; i < n; ++i) 
			myfile << i << "\t\t" << h_dist[i] <<"\n"; 
    	myfile.close();
  	}
  	else cout << "Unable to open file";

	ofstream myfile_path ("bf_path.txt");
	if (myfile_path.is_open())
	{
		vector<int> path;
		int current = goal;
		while(current != src)
		{
			path.push_back(current);
			current = h_came_from[current];
		}
		path.push_back(src);
		reverse(path.begin(), path.end());

		for (vector<int>::iterator i = path.begin(); i < path.end(); ++i)
		{
			myfile_path << *i << "\t\t";
		}
		myfile_path.close();
		
		int total = 0;
		for (vector<int>::iterator i = path.begin(); i < path.end()-1;)
		{
			int u = *i;
			int v = *(++i);
			int weight = h_weights[convert_dimension_2D_1D(u, v, n)];
			total += weight;
			cout << "u: " << u << ", v: " << v <<  ", weight: " << weight << "\n";
		}
		cout << "total: " << total <<"\n";
	} 
  	else cout << "Unable to open file";

	auto duration = duration_cast<milliseconds>(stop - start);
	cout << "duration :" << duration.count() << endl;
} 

void create_weights(int weights[], int n)
{
	for (int i = 0; i < n * n; i++) 
	{
		weights[i] = INF;
	}

	fstream fin;
	fin.open(fin_str, ios::in);

	vector<int> row;
	string line, word;
	getline(fin,line);

	while (!fin.eof())
	{
		row.clear();
		getline(fin, line);
		stringstream s(line);

		while (getline(s, word, ','))  
		{
			row.push_back(stoi(word));
		}

		weights[convert_dimension_2D_1D(row[0]-1, row[1]-1, n)] = row[2];
		weights[convert_dimension_2D_1D(row[1]-1, row[0]-1, n)] = row[2];
	}
	fin.close();
}

// Driver program to test above functions 
int main()
{
	int N = 16456;
	int* mat = (int *)malloc(N * N * sizeof(int));

	create_weights(mat, N);

	// for(int i=0; i<N*N; i++)
	// {
	// 	cout << i << ": "<< mat[i] << "\n";
	// }
	// cout << "\n";

	BellmanFord(0, 2324, N, mat);

	return 0; 
} 
