// A C++ program for Dijkstra's shortest path algorithm. 
#include <bits/stdc++.h>
#include <chrono>
#include <fstream>
#include <omp.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

using namespace std;
using namespace std::chrono;

#define INF 2000000000

const string fin_str = "../matlab/gr_optimal_control_3rd_order.csv";

typedef pair<int, int> iPair;

__global__ void bf(int n, int u, int const* d_weights, int* d_dist, bool* d_has_change, int* came_from)
{
	int v = blockIdx.x * blockDim.x + threadIdx.x;
	if(v < n)
	{
		d_has_change[v] = false;

		int weight = d_weights[u * n + v];
		if (weight < INF)
		{
			if (d_dist[v] > d_dist[u] + weight)
			{
				d_dist[v] = d_dist[u] + weight;
				d_has_change[v] = true;
				came_from[v] = u;
			}
		}
	}
}

//translate 2-dimension coordinate to 1-dimension
int convert_dimension_2D_1D(int x, int y, int n) 
{
	return x * n + y;
}

// The main function that finds shortest distances
void Dijkstra(int src, int goal, int n, int h_weights[]) 
{ 
	dim3 threadsPerBlock = 256;
	dim3 blocksPerGrid = ((n + threadsPerBlock.x - 1) / threadsPerBlock.x);
	
	// host 
	int *h_dist = (int *)calloc(sizeof(int), n);
	int *h_came_from = (int *)calloc(sizeof(int), n);
	bool *h_has_change = (bool *)calloc(sizeof(bool), n);
	
	for (int i=0; i<n; i++)
	{
		h_dist[i] = INF;
		h_came_from[i] = INF;
	}

	h_dist[src] = 0;
	h_came_from[src] = src;

	// device
	int* d_weights;
	int* d_dist;
	int* d_came_from;
	bool* d_has_change;

	hipMalloc(&d_weights, n * n * sizeof(int));
	hipMalloc(&d_dist, n * sizeof(int));
	hipMalloc(&d_came_from, n * sizeof(int));
	hipMalloc(&d_has_change, n * sizeof(bool));

	// copy host to device
	hipMemcpy(d_weights, h_weights, n * n * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_dist, h_dist, n * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_came_from, h_came_from, n * sizeof(int), hipMemcpyHostToDevice);

	priority_queue< iPair, vector <iPair> , greater<iPair> > pq; 
	pq.push(make_pair(0, src));

	int counter = 0;
	// main loop
	auto start = high_resolution_clock::now();
	while(!pq.empty())
	{
		int u = pq.top().second; 
		pq.pop();

		if(u == goal)
		{
			break;
		}

		counter++;

        // invoke kernel
		bf <<<blocksPerGrid, threadsPerBlock>>>(n, u, d_weights, d_dist, d_has_change, d_came_from);
	
		hipMemcpy(h_has_change, d_has_change, n * sizeof(bool), hipMemcpyDeviceToHost);
		hipMemcpy(h_dist, d_dist, sizeof(int) * n, hipMemcpyDeviceToHost);

		for (int i = 0; i < n; i++)
		{
			if (h_has_change[i])
			{
				pq.push(make_pair(h_dist[i], i)); 
			}
		}
	}

	auto stop = high_resolution_clock::now(); 

	cout << "counter: " << counter << "\n";

	hipMemcpy(h_came_from, d_came_from, n * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(d_weights);
	hipFree(d_dist);
	hipFree(d_came_from);
	hipFree(d_has_change);

	// Print shortest distances stored in dist[] 
	ofstream myfile ("dijkstra.txt");
  	if (myfile.is_open())
  	{
		for (int i = 0; i < n; ++i) 
			myfile << i << "\t\t" << h_dist[i] <<"\n"; 
    	myfile.close();
  	}
  	else cout << "Unable to open file";

	ofstream myfile_path ("dijkstra_path.txt");
	if (myfile_path.is_open())
	{
		vector<int> path;
		int current = goal;
		while(current != src)
		{
			path.push_back(current);
			current = h_came_from[current];
		}
		path.push_back(src);
		reverse(path.begin(), path.end());

		for (vector<int>::iterator i = path.begin(); i < path.end(); ++i)
		{
			myfile_path << *i << "\t\t";
		}
		myfile_path.close();
		
		int total = 0;
		for (vector<int>::iterator i = path.begin(); i < path.end()-1;)
		{
			int u = *i;
			int v = *(++i);
			int weight = h_weights[convert_dimension_2D_1D(u, v, n)];
			total += weight;
			cout << "u: " << u << ", v: " << v <<  ", weight: " << weight << "\n";
		}
		cout << "total: " << total <<"\n";
	} 
  	else cout << "Unable to open file";

	auto duration = duration_cast<milliseconds>(stop - start);
	cout << "duration :" << duration.count() << endl;
}

void create_weights(int weights[], int n)
{
	for (int i = 0; i < n * n; i++) 
	{
		weights[i] = INF;
	}

	fstream fin;
	fin.open(fin_str, ios::in);

	vector<int> row;
	string line, word;
	getline(fin,line);

	while (!fin.eof())
	{
		row.clear();
		getline(fin, line);
		stringstream s(line);

		while (getline(s, word, ','))
		{
			row.push_back(stoi(word));
		}

		weights[convert_dimension_2D_1D(row[0]-1, row[1]-1, n)] = row[2];
		weights[convert_dimension_2D_1D(row[1]-1, row[0]-1, n)] = row[2];
	}
	fin.close();
}

// Driver program to test above functions 
int main()
{
	int N = 16456;
	int* mat = (int *)malloc(N * N * sizeof(int));

	create_weights(mat, N);

	// for (int i=0; i< N*N; i++)
	// {
	// 	cout << mat[i] << " ";
	// }

	Dijkstra(0, 2324, N, mat);

	return 0; 
} 
